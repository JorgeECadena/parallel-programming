/*
 * Speed up = (22153.4 / 0.0136203) = 1,626,498.6821141972
 * Efficiency = (1,626,498.6821141972 / 512) = 3,176.7552385043
 * */
#include <iostream>
#include <chrono>
#include <cmath>
#include <hip/hip_runtime.h>

#define SIZE 5000000
#define N 10
#define THREADS 512
#define BLOCKS std::min(4, ((SIZE / THREADS) + 1))

using namespace std::chrono;

__device__ bool isPrime(int n) {
	if (n < 2) return false;

	for (int i = 2; i <= sqrtf(n); i++) {
		if (n % i == 0) return false;
	}

	return true;
}

__global__ void sumPrimes(long *counts) {
	__shared__ long cache[THREADS];

	int index = threadIdx.x + (blockIdx.x * blockDim.x);
	int cacheIdx = threadIdx.x;

	long acum;
	
	acum = 0;
	while (index < SIZE) {
		acum += isPrime(index) ? index : 0;
		index += blockDim.x * gridDim.x;
	}

	cache[cacheIdx] = acum;
	__syncthreads();

	int i = blockDim.x / 2;
	while (i > 0) {
		if (cacheIdx < i) {
			cache[cacheIdx] += cache[cacheIdx + i];
		}

		__syncthreads();
		i /= 2;
	}

	if (cacheIdx == 0) {
		counts[blockIdx.x] = cache[cacheIdx];
	}
}

int main() {
	long count;
	long counts[BLOCKS] = {0};
	long *deviceCounts;

	high_resolution_clock::time_point start, end;
	double timeElapsed;

	hipMalloc((void**) &deviceCounts, BLOCKS * sizeof(long));

	timeElapsed = 0;
	for (int j = 0; j < N; j++) {
		start = high_resolution_clock::now();

		sumPrimes<<<BLOCKS, THREADS>>>(deviceCounts);

		end = high_resolution_clock::now();
		timeElapsed += duration<double, std::milli>(end - start).count();
	}

	hipMemcpy(counts, deviceCounts, BLOCKS * sizeof(long), hipMemcpyDeviceToHost);

	count = 0;
	for (int i = 0; i < BLOCKS; i++) {
		count += counts[i];
	}

	std::cout << "Sum: " << count << "\n";
	std::cout << "Average execution time: " << timeElapsed / N << "ms\n";

	hipFree(deviceCounts);

	return 0;
}
